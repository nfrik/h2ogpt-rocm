#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.


#include <hip/hip_runtime.h>
#include "ops.cuh"
#include "kernels.cuh"
// #include <hipcub/device/device_scan.cuh>
#include <limits>
// #include <BinSearch.h>
#include <AAlloc.h>
#include <BinAlgo.h>
#include <cassert>
// #include <common.h>

using namespace BinSearch;
using std::cout;
using std::endl;

void histogramScatterAdd2D(float* histogram, int *index1, int *index2, float *src, int maxidx1, int n)
{
  int threads = 512;
  int num_blocks = n/threads;
  num_blocks = n % threads == 0 ? num_blocks : num_blocks + 1;
  kHistogramScatterAdd2D<<<num_blocks, 512>>>(histogram, index1, index2, src, maxidx1, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<num_blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int num_blocks = n/1024;
  num_blocks = n % 1024 == 0 ? num_blocks : num_blocks + 1;
  kQuantize<<<num_blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n)
{
  int num_blocks = n/1024;
  num_blocks = n % 1024 == 0 ? num_blocks : num_blocks + 1;
  kDequantize<<<num_blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, int blocksize, const int n)
{
  int num_blocks = n/blocksize;
  num_blocks = n % blocksize == 0 ? num_blocks : num_blocks + 1;
  if(STOCHASTIC == 1)
    assert(blocksize == 4096);

  if(blocksize == 4096)
    kQuantizeBlockwise<T, 4096, 4, STOCHASTIC><<<num_blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 2048)
    kQuantizeBlockwise<T, 2048, 4, 0><<<num_blocks, 512>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 1024)
    kQuantizeBlockwise<T, 1024, 4, 0><<<num_blocks, 256>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 512)
    kQuantizeBlockwise<T, 512, 2, 0><<<num_blocks, 256>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 256)
    kQuantizeBlockwise<T, 256, 2, 0><<<num_blocks, 128>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 128)
    kQuantizeBlockwise<T, 128, 2, 0><<<num_blocks, 64>>>(code, A, absmax, out, rand, rand_offset, n);
  else if(blocksize == 64)
    kQuantizeBlockwise<T, 64, 1, 0><<<num_blocks, 64>>>(code, A, absmax, out, rand, rand_offset, n);


  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n)
{
  int num_blocks = n/blocksize;
  num_blocks = n % blocksize == 0 ? num_blocks : num_blocks + 1;
  if(blocksize == 4096)
    kDequantizeBlockwise<T, 4096, 1024, 4><<<num_blocks, 4096/4>>>(code, A, absmax, out, n);
  else if(blocksize == 2048)
    kDequantizeBlockwise<T, 2048, 512, 4><<<num_blocks, 2048/4>>>(code, A, absmax, out, n);
  else if(blocksize == 1024)
    kDequantizeBlockwise<T, 1024, 256, 4><<<num_blocks, 1024/4>>>(code, A, absmax, out, n);
  else if(blocksize == 512)
    kDequantizeBlockwise<T, 512, 256, 2><<<num_blocks, 512/2>>>(code, A, absmax, out, n);
  else if(blocksize == 256)
    kDequantizeBlockwise<T, 256, 128, 2><<<num_blocks, 256/2>>>(code, A, absmax, out, n);
  else if(blocksize == 128)
    kDequantizeBlockwise<T, 128, 64, 2><<<num_blocks, 128/2>>>(code, A, absmax, out, n);
  else if(blocksize == 64)
    kDequantizeBlockwise<T, 64, 64, 1><<<num_blocks, 64/1>>>(code, A, absmax, out, n);

  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p,
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, bool skip_zeros, const int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
      if(max_unorm > 0.0f)
			{
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<num_blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<num_blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:

      if(max_unorm > 0.0f)
			{
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<num_blocks, 512>>>(g, p, state1, unorm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr,
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int num_blocks = n/4096;
  num_blocks = n % 4096 == 0 ? num_blocks : num_blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<num_blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<num_blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 256>>>(p, g, state1, unorm, beta1, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<num_blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 2048
#define NUM_2STATE 8
#define BLOCKSIZE_1STATE 2048
#define NUM_1STATE 8

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(T* p, T* g,
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, bool skip_zeros, int n)
{

	int num_blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
			num_blocks = n/BLOCKSIZE_2STATE;
			num_blocks = n % BLOCKSIZE_2STATE == 0 ? num_blocks : num_blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<num_blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(p, g, state1, state2, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale, skip_zeros, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
    case ADAGRAD:
			num_blocks = n/BLOCKSIZE_1STATE;
			num_blocks = n % BLOCKSIZE_1STATE == 0 ? num_blocks : num_blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<num_blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, skip_zeros, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int num_blocks = n/2048;
  num_blocks = n % 2048 == 0 ? num_blocks : num_blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<num_blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{
  cout << "" << endl;
  cout << "=============================================" << endl;
  cout << "ERROR: Your GPU does not support Int8 Matmul!" << endl;
  cout << "=============================================" << endl;
  cout << "" << endl;
  assert(false);

	return ;
  // const int falpha = 1;
  // const int fbeta = 0;
  // const void * alpha = &falpha;
  // const void * beta = &fbeta;
	// hipblasStatus_t status;

	// 		status = hipblasGemmEx(context->m_handle,
	// 				transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
	// 				transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
	// 				m, n,	k,
	// 				alpha, A, HIPBLAS_R_8I, lda, B, HIPBLAS_R_8I, ldb, beta,
	// 				C, HIPBLAS_R_32I, ldc,
  //         HIPBLAS_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  //   if (status != HIPBLAS_STATUS_SUCCESS)
  //   {
  //     std::cout << "CUBLAS ERROR: Status " << status << std::endl;
  //   }

}

void strided_gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc,
                    long long int strideA, long long int strideB, long long int strideC, int batchCount)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

  //cout << transposeA << transposeB << endl;
  //printf("%i %i %i\n", m,n,k);
  //printf("%i %i %i\n", lda,ldb,ldc);
  //printf("%i %i %i\n", strideA, strideB, strideC);
  //printf("%i\n", batchCount);

			status = hipblasGemmStridedBatchedEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIPBLAS_R_8I, lda, (long long int)strideA, B, HIPBLAS_R_8I, ldb, (long long int)strideB, beta,
					C, HIPBLAS_R_32I, ldc, (long long int)strideC, batchCount,
          HIPBLAS_R_32I, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}


template<int ORDER> int get_leading_dim(int dim1, int dim2)
{
	switch(ORDER)
	{
		case ROW:
      return dim2;
			break;
    case COL:
      return dim1;
      break;
    case COL32:
      // 32*row tiles
      return dim1*32;
      break;
    case COL_TURING:
      return 32*roundoff(dim1, 8);
      break;
    case COL_AMPERE:
      // 32*32 tiles
      return 32*roundoff(dim1, 32);
      break;
		default:
			return 0;
			break;
  }
}

template int get_leading_dim<ROW>(int dim1, int dim2);
template int get_leading_dim<COL>(int dim1, int dim2);
template int get_leading_dim<COL32>(int dim1, int dim2);

template <typename T, int SRC, int TARGET, bool transpose, int DTYPE> void transform(hipblasLtHandle_t ltHandle, T *A, T *out, int dim1, int dim2)
{
  cout << "" << endl;
  cout << "=============================================" << endl;
  cout << "ERROR: Your GPU does not support Int8 Matmul!" << endl;
  cout << "=============================================" << endl;
  cout << "" << endl;
  assert(false);
}

template void transform<int8_t, ROW, COL, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL32, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, ROW, COL32, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_TURING, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_AMPERE, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, COL32, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, COL32, ROW, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);

template <int FORMATB, int DTYPE_OUT, int SCALE_ROWS> int igemmlt(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc)
{
  cout << "" << endl;
  cout << "=============================================" << endl;
  cout << "ERROR: Your GPU does not support Int8 Matmul!" << endl;
  cout << "=============================================" << endl;
  cout << "" << endl;
  assert(false);

	return 0;
}

int fill_up_to_nearest_multiple(int value, int multiple)
{
  return value + (value % multiple == 0 ? 0 : (multiple - (value % multiple)));
}

void dequant_mm_int32_fp16(int *A, float *rowStats, float *colStats, half *out, float* newRowStats, float* newcolStats, half *bias, int numRows, int numCols)
{
  int threads = 512;
  int tileCols = fill_up_to_nearest_multiple(numCols, 32);
  int n = numRows*tileCols;
  int subtile_rows = 128;
  int tilesize = 32*subtile_rows;
  int num_blocks = numRows/subtile_rows;
  num_blocks += (numRows % subtile_rows == 0) ? 0 : 1;
  num_blocks = num_blocks*(tileCols/32);
  assert(threads <= tilesize);

  kdequant_mm_int32_fp16<4, 128, 512><<<num_blocks, threads>>>(A, rowStats, colStats, out, newRowStats, newcolStats, bias, numRows, numCols, tileCols, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define STATS_THREADS 64
#define STATS_ITEMS 4
#define STATS_ROWS 16
void getColRowStats(half * A, float *rowStats, float *colStats, int *nnz_count_row, float nnz_threshold, int rows, int cols)
{
  int tile_cols = STATS_THREADS*STATS_ITEMS;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, STATS_ROWS);
	int row_tiles = (tiledRows/STATS_ROWS);
	int col_tiles = (tiledCols/tile_cols);
	row_tiles = row_tiles > 0 ? row_tiles : 1;
	col_tiles = col_tiles > 0 ? col_tiles : 1;
  int num_blocks = row_tiles * col_tiles;

  if(nnz_threshold == 0.0)
    kgetColRowStats<half, STATS_THREADS, STATS_ITEMS, STATS_ROWS, STATS_THREADS*STATS_ITEMS, 0><<<num_blocks, STATS_THREADS>>>(A, rowStats, colStats, nnz_count_row, nnz_threshold, rows, cols, tiledRows, tiledCols);
  else if(nnz_threshold != 0.0)
    kgetColRowStats<half, STATS_THREADS, STATS_ITEMS, STATS_ROWS, STATS_THREADS*STATS_ITEMS, 1><<<num_blocks, STATS_THREADS>>>(A, rowStats, colStats, nnz_count_row, nnz_threshold, rows, cols, tiledRows, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());

}

void doubleRowColQuant(half * A, float *rowStats, float *colStats, char *out_col_normed, char *out_row_normed, int *rowidx, int *colidx, half *val, int *nnz_block_ptr, float threshold, int rows, int cols)
{
  int threads = 64;
  int items_per_thread = 4;
  int tile_cols = threads*items_per_thread;
  int tile_rows = 16;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
	int row_tiles = (tiledRows/tile_rows);
	int col_tiles = (tiledCols/tile_cols);
	row_tiles = row_tiles > 0 ? row_tiles : 1;
	col_tiles = col_tiles > 0 ? col_tiles : 1;
  int num_blocks = row_tiles * col_tiles;


  if(threshold > 0.0f)
    kDoubleRowColQuant<64, 4, 16, 64*4, 1><<<num_blocks, threads>>>(A, rowStats, colStats, out_col_normed, out_row_normed, rowidx, colidx, val, nnz_block_ptr, threshold, rows, cols, tiledCols);
  else
    kDoubleRowColQuant<64, 4, 16, 64*4, 0><<<num_blocks, threads>>>(A, rowStats, colStats, out_col_normed, out_row_normed, rowidx, colidx, val, nnz_block_ptr, threshold, rows, cols, tiledCols);

  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <int FORMAT, int TRANSPOSE> void transformRowToFormat(char * A, char *out, int rows, int cols)
{
  int threads = 256;
  int items_per_thread = 8;
  // we load 128 column values per warp
  int tile_cols = 32*items_per_thread;
  int tile_rows = 32;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
	int row_tiles = (tiledRows/tile_rows);
	int col_tiles = (tiledCols/tile_cols);
	row_tiles = row_tiles > 0 ? row_tiles : 1;
	col_tiles = col_tiles > 0 ? col_tiles : 1;
  int num_blocks = row_tiles * col_tiles;

  int outCols = fill_up_to_nearest_multiple(cols, 32);
  int outRows = fill_up_to_nearest_multiple(rows, 32);
  if(FORMAT == COL_TURING)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 8);
    else
      outRows = fill_up_to_nearest_multiple(rows, 8);
  }
  else if(FORMAT == COL_AMPERE)
  {
    if(TRANSPOSE)
      outRows = fill_up_to_nearest_multiple(cols, 32);
    else
      outRows = fill_up_to_nearest_multiple(rows, 32);
  }
  else
  {
    if(TRANSPOSE)
    {
      outCols = fill_up_to_nearest_multiple(rows, 32);
      outRows = cols;
    }
  }

  kTransformRowToFormat<256, 8, 32, 32*8, TRANSPOSE, FORMAT><<<num_blocks, threads>>>(A, out, rows, cols, tiledCols, outRows, outCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void spmm_coo(hipsparseHandle_t handle, int *A_rowidx, int *A_colidx, half *A_vals, int A_nnz, int A_rows, int A_cols, int B_cols, int ldb, half *B, int ldc, half* C, bool transposed_B)
{

  cout << "" << endl;
  cout << "=============================================" << endl;
  cout << "ERROR: Your GPU does not support Int8 Matmul!" << endl;
  cout << "=============================================" << endl;
  cout << "" << endl;
  assert(false);

	return;
}

template <typename T, int BITS> void spmm_coo_very_sparse_naive(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, T *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB)
{

  kspmm_coo_very_sparse_naive<T, 8, BITS><<<nnz_rows, 256>>>(max_count, max_idx, offset_rowidx, rowidx, colidx, values, B, out, dequant_stats, nnz, rowsA, rowsB, colsB);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}


template <int FORMAT> void extractOutliers(char * A, int *idx, char *out, int idx_size, int rows, int cols)
{
  int threads = 256;
  // we load 128 column values per warp
  int tiledCols = tiledCols = fill_up_to_nearest_multiple(cols, 32);
  int tiledRows = 0;

	int num_blocks = idx_size;

  if(FORMAT == COL_TURING)
  {
      tiledRows = fill_up_to_nearest_multiple(rows, 8);
  }
  else if(FORMAT == COL_AMPERE)
  {
      tiledRows = fill_up_to_nearest_multiple(rows, 32);
	}

  kExtractOutliers<FORMAT><<<num_blocks, threads>>>(A, idx, out, idx_size, rows, cols, tiledRows, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void extractOutliers<COL_TURING>(char * A, int *idx, char *out, int idx_size, int rows, int cols);
template void extractOutliers<COL_AMPERE>(char * A, int *idx, char *out, int idx_size, int rows, int cols);

template void spmm_coo_very_sparse_naive<half, 16>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, half *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);
template void spmm_coo_very_sparse_naive<signed char, 8>(int *max_count, int *max_idx, int *offset_rowidx, int *rowidx, int *colidx, half *values, signed char *B, half *out, float *dequant_stats, int nnz_rows, int nnz, int rowsA, int rowsB, int colsB);

template int igemmlt<COL_TURING, 32, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_TURING, 8, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_TURING, 8, 1>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 32, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 8, 0>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);
template int igemmlt<COL_AMPERE, 8, 1>(hipblasLtHandle_t ltHandle, int m, int n, int k, const int8_t *A, const int8_t *B, void *C, float *row_scale, int lda, int ldb, int ldc);

template void transformRowToFormat<COL32, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL32, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_TURING, 1>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 0>(char * A, char *out, int rows, int cols);
template void transformRowToFormat<COL_AMPERE, 1>(char * A, char *out, int rows, int cols);

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 0>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 0>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<half, 1>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void quantizeBlockwise<float, 1>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, int blocksize, const int n);
template void dequantizeBlockwise<half>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n);
template void dequantizeBlockwise<float>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n);

#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)
MAKE_optimizer32bit(ADAGRAD, half)
MAKE_optimizer32bit(ADAGRAD, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)

#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, bool skip_zeros, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(half, ADAGRAD);
MAKE_optimizerStatic8bitBlockwise(float, ADAGRAD);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);
